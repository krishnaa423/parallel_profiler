#include "hip/hip_runtime.h"
// Recommended compile (works on many systems):
//   nvcc -O3 -arch=sm_60 -ccbin mpicxx -o mpi_cuda_dot mpi_cuda_dot.cu
// Alternatives (cluster-dependent):
//   mpicxx -O3 mpi_cuda_dot.cu -lcudart -L$CUDA_HOME/lib64 -o mpi_cuda_dot
// Run: mpirun -np 4 ./mpi_cuda_dot 100000000

#include <cstdio>
#include <cstdlib>
#include <mpi.h>
#include <hip/hip_runtime.h>

__global__ void dot_kernel(const double* __restrict__ A,
                           const double* __restrict__ B,
                           double* sum, long long n, long long start_idx) {
    double local = 0.0;
    long long gid = blockIdx.x * (long long)blockDim.x + threadIdx.x;
    long long stride = (long long)blockDim.x * gridDim.x;
    for (long long k = gid; k < n; k += stride) {
        long long g = start_idx + k;
        double a = (double)g;
        double b = 1.0 / (double)(g + 1);
        local += a * b;
    }
    atomicAdd(sum, local);
}

static inline void check(hipError_t e, const char* msg) {
    if (e != hipSuccess) { fprintf(stderr, "%s: %s\n", msg, hipGetErrorString(e)); MPI_Abort(MPI_COMM_WORLD, 2); }
}

int main(int argc, char** argv) {
    MPI_Init(&argc, &argv);
    int rank=0, size=1; MPI_Comm_rank(MPI_COMM_WORLD, &rank); MPI_Comm_size(MPI_COMM_WORLD, &size);

    if (argc < 2) {
        if (rank==0) fprintf(stderr, "usage: %s n\n", argv[0]);
        MPI_Finalize(); return 1;
    }
    long long n = atoll(argv[1]);
    if (n <= 0) { if (rank==0) fprintf(stderr, "n must be > 0\n"); MPI_Finalize(); return 1; }

    // Decompose global range
    long long chunk = n / size;
    long long r = n % size;
    long long local_n = chunk + (rank < r ? 1 : 0);
    long long start = rank * chunk + (rank < r ? rank : r);

    // Pick a GPU per rank (round-robin)
    int devCount = 0; hipGetDeviceCount(&devCount);
    if (devCount == 0) { if (rank==0) fprintf(stderr, "No CUDA devices found\n"); MPI_Abort(MPI_COMM_WORLD, 3); }
    int dev = rank % devCount;
    check(hipSetDevice(dev), "set device");

    // Device accumulator
    double *dSum = nullptr;
    check(hipMalloc(&dSum, sizeof(double)), "hipMalloc dSum");
    check(hipMemset(dSum, 0, sizeof(double)), "memset dSum");

    // Time
    hipEvent_t t0, t1; hipEventCreate(&t0); hipEventCreate(&t1);
    hipEventRecord(t0);

    // Launch: compute A[i], B[i] on the fly on device to avoid H2D copies
    int tpb = 256;
    int maxBlocks = 1024;
    int blocks = (int)((local_n + tpb - 1) / tpb);
    if (blocks > maxBlocks) blocks = maxBlocks;

    dot_kernel<<<blocks, tpb>>>(nullptr, nullptr, dSum, local_n, start);
    check(hipGetLastError(), "kernel");
    check(hipDeviceSynchronize(), "sync");

    hipEventRecord(t1); hipEventSynchronize(t1);
    float ms = 0.0f; hipEventElapsedTime(&ms, t0, t1);

    double local_sum = 0.0, global_sum = 0.0;
    check(hipMemcpy(&local_sum, dSum, sizeof(double), hipMemcpyDeviceToHost), "D2H sum");
    MPI_Reduce(&local_sum, &global_sum, 1, MPI_DOUBLE, MPI_SUM, 0, MPI_COMM_WORLD);

    if (rank == 0) {
        printf("[MPI+CUDA] n=%lld ranks=%d (gpu/devCount may vary)  sum=%.12f  time=%.3f s\n",
               n, size, global_sum, ms*1e-3);
    }

    hipFree(dSum);
    MPI_Finalize();
    return 0;
}

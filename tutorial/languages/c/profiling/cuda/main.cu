// Compile: nvcc -O3 -arch=sm_60 -o cuda_dot cuda_dot.cu
// Run:     ./cuda_dot 100000000
// Note: uses atomicAdd on double (needs sm_60+). Change to float if needed.

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

__global__ void dot_kernel(const double* __restrict__ A,
                           const double* __restrict__ B,
                           double* sum, long long n) {
    double local = 0.0;
    long long gid = blockIdx.x * (long long)blockDim.x + threadIdx.x;
    long long stride = (long long)blockDim.x * gridDim.x;
    for (long long i = gid; i < n; i += stride) {
        local += A[i] * B[i];
    }
    atomicAdd(sum, local);
}

static inline void check(hipError_t e, const char* msg) {
    if (e != hipSuccess) { fprintf(stderr, "%s: %s\n", msg, hipGetErrorString(e)); exit(2); }
}

static inline double now_sec() {
    hipDeviceSynchronize();
    static hipEvent_t start, stop;
    static bool inited = false;
    if (!inited) { hipEventCreate(&start); hipEventCreate(&stop); inited = true; }
    hipEventRecord(start, 0);
    hipEventRecord(stop, 0); // just to create valid events
    return 0.0; // not used
}

int main(int argc, char** argv) {
    if (argc < 2) { fprintf(stderr, "usage: %s n\n", argv[0]); return 1; }
    long long n = atoll(argv[1]);
    if (n <= 0) { fprintf(stderr, "n must be > 0\n"); return 1; }

    double* hA = (double*)malloc(sizeof(double)*n);
    double* hB = (double*)malloc(sizeof(double)*n);
    if (!hA || !hB) { fprintf(stderr, "host allocation failed\n"); return 2; }
    for (long long i = 0; i < n; ++i) { hA[i] = (double)i; hB[i] = 1.0/(double)(i+1); }

    double *dA = nullptr, *dB = nullptr, *dSum = nullptr;
    check(hipMalloc(&dA, n*sizeof(double)), "hipMalloc dA");
    check(hipMalloc(&dB, n*sizeof(double)), "hipMalloc dB");
    check(hipMalloc(&dSum, sizeof(double)), "hipMalloc dSum");
    check(hipMemcpy(dA, hA, n*sizeof(double), hipMemcpyHostToDevice), "H2D A");
    check(hipMemcpy(dB, hB, n*sizeof(double), hipMemcpyHostToDevice), "H2D B");
    check(hipMemset(dSum, 0, sizeof(double)), "Memset sum");

    hipEvent_t t0, t1;
    hipEventCreate(&t0); hipEventCreate(&t1);
    hipEventRecord(t0);

    int tpb = 256;
    int maxBlocks = 1024;
    int blocks = (int)((n + tpb - 1) / tpb);
    if (blocks > maxBlocks) blocks = maxBlocks;

    dot_kernel<<<blocks, tpb>>>(dA, dB, dSum, n);
    check(hipGetLastError(), "kernel launch");
    check(hipDeviceSynchronize(), "sync");

    hipEventRecord(t1); hipEventSynchronize(t1);
    float ms = 0.0f; hipEventElapsedTime(&ms, t0, t1);

    double sum = 0.0;
    check(hipMemcpy(&sum, dSum, sizeof(double), hipMemcpyDeviceToHost), "D2H sum");

    printf("[CUDA] n=%lld  sum=%.12f  time=%.3f s\n", n, sum, ms*1e-3);

    hipFree(dA); hipFree(dB); hipFree(dSum);
    free(hA); free(hB);
    return 0;
}
